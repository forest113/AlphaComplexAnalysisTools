#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <gmp.h>
#include "cgbn/cgbn.h"
#include <inttypes.h>


#define TPI 32
#define BITS 1024

typedef cgbn_context_t<TPI>         context_t;
typedef cgbn_env_t<context_t, BITS> env_t;

class MP_float {
   private:
      cgbn_mem_t<1024> num;
      cgbn_mem_t<1024> denom;
      cgbn_error_report_t *report;

   public:   
      MP_float(float a) {
          for(int i=0; i<(1024+31)/32; i++){
              this->num._limbs[i] = 0;
              this->denom._limbs[i] = 0;
          }
          int sign = 0;
          uint32_t value = 0;
          uint32_t exp = 1;
          if(a < 0){
              sign = 1;
          }
          float temp = a;

          int temp_int = temp;
          float mod = temp - temp_int;
          while(mod > 0){
              temp *= 10;
              temp_int = temp;
              mod = temp - temp_int;
              exp *= 10;
          }
          if(sign){
              value = -temp;
          }
          else{
              value = temp;
          }
          context_t bn_context(cgbn_report_monitor, this->report, 0);                                 // create a CGBN context
          env_t bn_env(bn_context); 
          env_t::cgbn_t numerator, denom, num_float;
          env_t::cgbn_t num_float_signed;
          cgbn_set_ui32(bn_env, num_float, 1);
          cgbn_set_ui32(bn_env, numerator, value);
          cgbn_set_ui32(bn_env, denom, exp);
          printf("*********num:%d,denom:%d\n",cgbn_get_ui32(bn_env, numerator),cgbn_get_ui32(bn_env,denom));
          //bn_env.div(num_float, numerator, denom);
          if(sign){
              bn_env.negate(num_float_signed, numerator);
              cgbn_store(bn_env, &(this->num), num_float_signed);
              

          }
          else{
              cgbn_store(bn_env, &(this->num), numerator);
          }
          cgbn_store(bn_env, &(this->denom), denom);
      }
      
      MP_float operator+ (MP_float num1) {
          MP_float result(0.0);
          context_t bn_context(cgbn_report_monitor, this->report, 0);                                 // create a CGBN context
          env_t bn_env(bn_context);                       // construct a bn environment for 1024 bit math
          env_t::cgbn_t a,a_denom, b, b_denom, r, const_10;
          cgbn_set_ui32(bn_env, const_10, 10);
          bn_env.load(a, &(this->num));
          bn_env.load(a_denom, &(this->denom));

          bn_env.load(b, &(num1.num));
          bn_env.load(b_denom, &(num1.denom));
          
          int a_denom_greater = bn_env.compare(a_denom, b_denom);
          while(bn_env.compare(a_denom, b_denom) != 0){
              //printf("hi");
              if(a_denom_greater == 1){
                  bn_env.mul(b_denom, b_denom, const_10);
                  bn_env.mul(b, b, const_10);
              }
              else{
                  bn_env.mul(a_denom, a_denom, const_10);
                  bn_env.mul(a, a, const_10);
              }
          }
          
          bn_env.add(r, a, b);
          bn_env.store((&result.num), r);
          bn_env.store((&result.denom), b_denom);

          return result;
      }
      
      MP_float operator- (MP_float num1) {
          MP_float result(0.0);
          context_t bn_context(cgbn_report_monitor, this->report, 0);                                 // create a CGBN context
          env_t bn_env(bn_context);                       // construct a bn environment for 1024 bit math
          env_t::cgbn_t a,a_denom, b, b_denom, r, const_10;
          cgbn_set_ui32(bn_env, const_10, 10);
          bn_env.load(a, &(this->num));
          bn_env.load(a_denom, &(this->denom));

          bn_env.load(b, &(num1.num));
          bn_env.load(b_denom, &(num1.denom));
          
          int a_denom_greater = bn_env.compare(a_denom, b_denom);
          while(bn_env.compare(a_denom, b_denom) != 0){
              if(a_denom_greater){
                  bn_env.mul(b_denom, b_denom, const_10);
                  bn_env.mul(b, b, const_10);
              }
              else{
                  bn_env.mul(a_denom, a_denom, const_10);
                  bn_env.mul(a, a, const_10);
              }
          }
          
          bn_env.sub(r, a, b);
          bn_env.store((&result.num), r);
          bn_env.store((&result.denom), b_denom);

          return result;
      }
      
      MP_float operator* (MP_float num1) {
          MP_float result(0.0);
          context_t bn_context(cgbn_report_monitor, this->report, 0);                                 // create a CGBN context
          env_t bn_env(bn_context);                       // construct a bn environment for 1024 bit math
          env_t::cgbn_t a,a_denom, b, b_denom, r, r_denom, const_10;
          cgbn_set_ui32(bn_env, const_10, 10);
          bn_env.load(a, &(this->num));
          bn_env.load(a_denom, &(this->denom));

          bn_env.load(b, &(num1.num));
          bn_env.load(b_denom, &(num1.denom));
          
          bn_env.mul(r, a, b);
          bn_env.mul(r_denom, a_denom, b_denom);
          bn_env.store((&result.num), r);
          bn_env.store((&result.denom), r_denom);

          return result;
      }
      
      uint32_t* get_limbs(){
          return this->num._limbs;
      }
      
      double get_float(){
          context_t bn_context(cgbn_report_monitor, this->report, 0);                                 // create a CGBN context
          env_t bn_env(bn_context);                       // construct a bn environment for 1024 bit math
          env_t::cgbn_t double_num, double_denom, const_10;
          cgbn_set_ui32(bn_env, const_10, 10);
          
          while((this->num._limbs[1]!=0 && (int)this->num._limbs[1]!=-1) && (this->denom._limbs[2]>0)){
              bn_env.load(double_num, &(this->num));
              bn_env.load(double_denom, &(this->denom));

              bn_env.div(double_num, double_num, const_10);
              bn_env.div(double_denom, double_denom, const_10);
              bn_env.store(&(this->num), double_num);
              bn_env.store(&(this->denom), double_denom);
          }
          uint64_t num = 0;
          num += this->num._limbs[1];
          num = num>>32;
          num += this->num._limbs[0];
          uint64_t denom = 0;
          denom += this->denom._limbs[1];
          denom = denom>>32;
          denom += this->denom._limbs[0];
          double result = (long double) num/(long double) denom;
          return result;

      }
      
};


int main(){
    MP_float A(10.0);
    MP_float B(10.1);
    MP_float sum(0);
    sum = A + B;
    uint32_t *limbs = sum.get_limbs();

    for(int i=0; i<(1024+31)/32; i++){
        printf("%d ",limbs[i]);
    }

    /*for(int j=0; j<1; j++){
        sum = sum+A;
        uint32_t *limbs = sum.get_limbs();
        for(int i=0; i<(1024+31)/32; i++){
            printf("%08X ",limbs[i]);
        }
    }*/
    printf("answerrr :%f ",sum.get_float());

    //std::cout<<std:endl;
}


